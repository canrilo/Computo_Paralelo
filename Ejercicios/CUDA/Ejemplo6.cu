
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void kernel(int *d, int n){
	__shared__ int s[64];
	int t =threadIdx.x;
	int tr=n-t-1;
	s[t]=d[t];
	__syncthreads();
	d[t]=s[tr];
}

int main (void)
{
	const int n=64;
	int a[n],r[n],d[n];
	for(int i=0;i<n;i++)
	{
		a[i]=i;
		r[i]=n-i-1;
		d[i]=0;
	}
	
	int *d_d;
	hipMalloc(&d_d, n * sizeof(int));
	hipMemcpy(d_d, a, n*sizeof(int), hipMemcpyHostToDevice);
	kernel<<<1,n>>>(d_d, n);
	hipMemcpy(d, d_d, n*sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 0; i < n; i++)
		if (d[i] != r[i])
			printf("Verificar- Hay un error");
	
	printf("En teoria 3 deberia ser igual a %d\n",d[n-3-1]);
}
