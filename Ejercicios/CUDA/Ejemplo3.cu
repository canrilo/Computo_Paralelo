
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
//#define N 16384

__global__ void addCincoVec(int *a, int N)
	{
		int tid=threadIdx.x+blockIdx.x*blockDim.x;
		if(tid<N)
		{
			a[tid]=a[tid]+5;
		}
	}

int main (void)
{
	
	int *dev_a,*a;
	int N,num_blocs,num_hilos;
	float elapsedTime;
	printf("Ingrese el tamano del vector\n");
	scanf("%d",&N);
	//asignar memoria en la GPU
	a=(int *)malloc(N*sizeof(int));
	hipMalloc((void**)&dev_a,N*sizeof(int));
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	for(int i=0; i<N;i++)
	{
		a[i]=i;
	}
	if(N>1024){
		num_blocs=(int)(N/1024)+1;
		num_hilos=1024;
	}else{
		num_blocs=1;
		num_hilos=N;
	}
	//copiar el arreglo 'a' y 'b' en la GPU
	hipMemcpy(dev_a,a,N*sizeof(int),hipMemcpyHostToDevice);
	printf("Se van a ejecutar %d bloques con %d hilos\n",num_blocs,num_hilos);
	hipEventRecord(start,0);
	addCincoVec<<<num_blocs,num_hilos>>>(dev_a,N);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	printf("El tiempo tomado fue de %3.3f ms\n",elapsedTime);
	hipMemcpy(a,dev_a,N*sizeof(int),hipMemcpyDeviceToHost);
	printf("En el renglon 0: %d\n",a[0]);
	printf("En el renglon %d: %d\n",(int)N/2,a[(int(N/2))]);
	printf("En el renglon %d: %d\n",N-1,a[N-1]);
	free(a);
	hipFree(dev_a);
	return 0;
}