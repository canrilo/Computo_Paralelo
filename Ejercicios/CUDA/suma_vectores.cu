
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10

__global__ void addvec(int *a, int *b, int *c)
{
	int tid=threadIdx.x; // Manejar todos los datos con este índice
	if(tid<N)
	{
		c[tid]=a[tid]+b[tid];
	}
}

int main(void)
{
	int a[N], b[N], c[N],i;
	int *dev_a, *dev_b, *dev_c;
	
	//Asignación de espacio en la memoria de GPU
	hipMalloc((void**)&dev_a,N*sizeof(int));
	hipMalloc((void**)&dev_b,N*sizeof(int));
	hipMalloc((void**)&dev_c,N*sizeof(int));
	
	//Inicializar los datos originales en el CPU
	for (i = 0; i < N; i++)
	{
		a[i]=i*2;
		b[i]=i*2+1;
	}
	
	//Copia de vectores a la GPU
	hipMemcpy(dev_a,a,N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,N*sizeof(int),hipMemcpyHostToDevice);

	//Se lanza el kernel
	addvec<<<1,N>>>(dev_a,dev_b,dev_c);
	
	//Se recuperan los datos de la GPU
	hipMemcpy(c,dev_c,N*sizeof(int),hipMemcpyDeviceToHost);
	
	//Se muestra el resultado
	for (i = 0; i < N; i++)
	{
		printf("%d+%d=%d\n",a[i],b[i],c[i]);
	}
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	
	return 0;
	
}
