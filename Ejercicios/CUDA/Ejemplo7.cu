
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 256
__global__ void kernel(int *a, int *b, int *c){
	__shared__ int s[N];
	__shared__ int r[N];
	int t =threadIdx.x;
	r[t]=b[t];
	s[t]=a[t];
	__syncthreads();
	c[t]=s[t]+r[t];
}

int main (void)
{
	//const int n=64;
	int a[N],c[N],b[N],i;
	for(i=0;i<N;i++)
	{
		a[i]=i;
		b[i]=N-i-1;
		c[i]=0;
	}
	
	int *a_d,*b_d, *c_d;
	hipMalloc(&a_d, N * sizeof(int));
	hipMalloc(&b_d, N * sizeof(int));
	hipMalloc(&c_d, N * sizeof(int));
	hipMemcpy(a_d, a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(b_d, b, N*sizeof(int), hipMemcpyHostToDevice);
	kernel<<<1,N>>>(a_d, b_d,c_d);
	hipMemcpy(c, c_d, N*sizeof(int), hipMemcpyDeviceToHost);
	for (i = 1; i < N; i++)
		if (c[i] != c[i-1])
			printf("Verificar- Hay un error");
	
	hipFree(a_d);
	hipFree(b_d);
	hipFree(c_d);
}
