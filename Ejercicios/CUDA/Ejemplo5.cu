
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// Kernel that executes on the CUDA device

__global__ void square_array(float *a, float *b, int N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<N)
	{
		a[idx] = a[idx] * a[idx];
	} 
}

// main routine that executes on the host
int main(void)
{
	float elapsedTime;
	float *a, *a_d,*b,*b_d; // Pointer to host & device arrays
	const int N = 12; // Number of elements in arrays
	int ind=0,iteraciones=10;
	size_t size = N * sizeof(float);
	a = (float *)malloc(size); // Allocate array on host
	b = (float *)malloc(size); // Allocate array on host
	hipMalloc((void **) &a_d, size); // Allocate array on device
	hipMalloc((void **) &b_d, size); // Allocate array on device
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Initialize host array and copy it to CUDA device
	for (int i=0; i<N; i++)
	{
		a[i] = (float)i;
		b[i] = (float)i+1;
	}		
	hipMemcpy(a_d, a, size,hipMemcpyHostToDevice);
	hipMemcpy(b_d, b, size,hipMemcpyHostToDevice);
	// Do calculation on device:
	int block_size = 4;
	int n_blocks = N/block_size;
	hipEventRecord(start,0);
	while(ind<iteraciones)
	{
		square_array <<< n_blocks, block_size >>> (a_d,b_d, N);
		ind++;
	}
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	printf("El tiempo tomado para %d iteraciones fue de %3.3f ms\n",iteraciones,elapsedTime/10);
	// Retrieve result from device and store it in host array
	hipMemcpy(a, a_d, sizeof(float)*N,hipMemcpyDeviceToHost);
	/*// Print results
	for (int i=0; i<N; i++) 
		printf("%d %f\n", i, a[i]);
	*/
	// Cleanup
	free(a);
	free(b);
	hipFree(a_d);
	hipFree(b_d);
}