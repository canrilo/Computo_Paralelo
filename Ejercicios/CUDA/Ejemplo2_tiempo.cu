
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 16384

__global__ void addVecGrande(int *a, int *b, int *c)
	{
		int tid=threadIdx.x+blockIdx.x*blockDim.x;
		if(tid<N)
		{
			c[tid]=a[tid]+b[tid];
		}
	}

int main (void)
{
	
	int *dev_a, *dev_b, *dev_c,*a,*b,*c;
	float elapsedTime;
	//asignar memoria en la GPU
	a=(int *)malloc(N*sizeof(int));
	b=(int *)malloc(N*sizeof(int));
	c=(int *)malloc(N*sizeof(int));
	hipMalloc((void**)&dev_a,N*sizeof(int));
	hipMalloc((void**)&dev_b,N*sizeof(int));
	hipMalloc((void**)&dev_c,N*sizeof(int));
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	for(int i=0; i<N;i++)
	{
		a[i]=i;
		b[i]=i+1;
	}
	//copiar el arreglo 'a' y 'b' en la GPU
	hipMemcpy(dev_a,a,N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,N*sizeof(int),hipMemcpyHostToDevice);
	printf("Se van a ejecutar 128 bloques con 128 hilos\n");
	hipEventRecord(start,0);
	addVecGrande<<<128,128>>>(dev_a,dev_b,dev_c);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	printf("El tiempo tomado fue de %3.3f ms\n",elapsedTime);
	hipMemcpy(c,dev_c,N*sizeof(int),hipMemcpyDeviceToHost);
	printf("En el renglon 0: \t%d\t+\t%d\t=\t%d\n",a[0],b[0],c[0]);
	printf("En el renglon 10: \t%d\t+\t%d\t=\t%d\n",a[10],b[10],c[10]);
	printf("En el renglon %d: \t%d\t+\t%d\t=\t%d\n",N,a[N-1],b[N-1],c[N-1]);
	
	printf("Se van a ejecutar 256 bloques con 64 hilos\n");
	hipEventRecord(start,0);
	addVecGrande<<<256,64>>>(dev_a,dev_b,dev_c);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	printf("El tiempo tomado fue de %3.3f ms\n",elapsedTime);
	hipMemcpy(c,dev_c,N*sizeof(int),hipMemcpyDeviceToHost);
	printf("En el renglon 0: \t%d\t+\t%d\t=\t%d\n",a[0],b[0],c[0]);
	printf("En el renglon 10: \t%d\t+\t%d\t=\t%d\n",a[10],b[10],c[10]);
	printf("En el renglon %d: \t%d\t+\t%d\t=\t%d\n",N,a[N-1],b[N-1],c[N-1]);
	
	printf("Se van a ejecutar 32 bloques con 512 hilos\n");
	hipEventRecord(start,0);
	addVecGrande<<<32,512>>>(dev_a,dev_b,dev_c);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	printf("El tiempo tomado fue de %3.3f ms\n",elapsedTime);
	hipMemcpy(c,dev_c,N*sizeof(int),hipMemcpyDeviceToHost);
	printf("En el renglon 0: \t%d\t+\t%d\t=\t%d\n",a[0],b[0],c[0]);
	printf("En el renglon 10: \t%d\t+\t%d\t=\t%d\n",a[10],b[10],c[10]);
	printf("En el renglon %d: \t%d\t+\t%d\t=\t%d\n",N,a[N-1],b[N-1],c[N-1]);
	
	free(a);
	free(b);
	free(c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return 0;
}