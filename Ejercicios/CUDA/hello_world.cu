
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void holaCUDA(float e){
	
	printf("Hola, soy el hilo %d del bloque %d con valor pi -> %f\n", threadIdx.x, blockIdx.x,e);
}

int main (int argc, char **argv)
{
	holaCUDA<<<3,4>>>(3.1416);
	hipDeviceReset(); //Reinicializa el device
	return 0;	
}
