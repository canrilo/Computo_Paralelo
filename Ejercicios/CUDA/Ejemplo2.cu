
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
//#define N 16384

__global__ void addVecGrande(int *a, int *b, int *c, int N)
	{
		int tid=threadIdx.x+blockIdx.x*blockDim.x;
		if(tid<N)
		{
			c[tid]=a[tid]+b[tid];
		}
	}

int main (void)
{
	
	int *dev_a, *dev_b, *dev_c,*a,*b,*c;
	int N,num_blocs,num_hilos,div;
	printf("Ingrese el tamano del vector (multiplo de 32)\n");
	scanf("%d",&N);
	if(N%32!=0){
		printf("El numero debe ser multiplo de 32\n");
		exit(1);
	}
	div=N/32;
	//asignar memoria en la GPU
	a=(int *)malloc(N*sizeof(int));
	b=(int *)malloc(N*sizeof(int));
	c=(int *)malloc(N*sizeof(int));
	hipMalloc((void**)&dev_a,N*sizeof(int));
	hipMalloc((void**)&dev_b,N*sizeof(int));
	hipMalloc((void**)&dev_c,N*sizeof(int));
	
	for(int i=0; i<N;i++)
	{
		a[i]=i;
		b[i]=i+1;
	}
	if(div>128&&(div/4)%2==0)
	{
		num_blocs=128;
		num_hilos=N/128;
	} else{
		num_hilos=div;
		num_blocs=32;
	}
	//copiar el arreglo 'a' y 'b' en la GPU
	hipMemcpy(dev_a,a,N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,N*sizeof(int),hipMemcpyHostToDevice);
	printf("Se van a ejecutar %d bloques con %d hilos\n",num_blocs,num_hilos);
	addVecGrande<<<num_blocs,num_hilos>>>(dev_a,dev_b,dev_c,N);
	hipMemcpy(c,dev_c,N*sizeof(int),hipMemcpyDeviceToHost);
	printf("En el renglon 0: \t%d\t+\t%d\t=\t%d\n",a[0],b[0],c[0]);
	printf("En el renglon 10: \t%d\t+\t%d\t=\t%d\n",a[10],b[10],c[10]);
	printf("En el renglon %d: \t%d\t+\t%d\t=\t%d\n",N,a[N-1],b[N-1],c[N-1]);
	free(a);
	free(b);
	free(c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return 0;
}