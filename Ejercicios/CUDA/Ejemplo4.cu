
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void sumaMatrices(int *a, int *b, int *c, int N)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int fil = blockIdx.y * blockDim.y + threadIdx.y;
	int indice = fil * N + col;
	if(fil<N&&col<N)
	{
		c[indice]=a[indice]+b[indice];
	}
}

int main (void)
{
	
	
	int *dev_a, *dev_b, *dev_c,*a,*b,*c;
	int T,div=2, iteraciones=10,ind=0;
	int N,i,j;
	float elapsedTime;
	printf("Ingrese el tamano de las matrices\n");
	scanf("%d",&N);
	
	
	a=(int*)malloc(N*N*sizeof(int));
	b=(int*)malloc(N*N*sizeof(int));
	c=(int*)malloc(N*N*sizeof(int));
	hipMalloc((void**)&dev_a,N*N*sizeof(int));
	hipMalloc((void**)&dev_b,N*N*sizeof(int));
	hipMalloc((void**)&dev_c,N*N*sizeof(int));
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
		{
			a[i*N+j]=i*N+j;
			b[i*N+j]=i*N+j;
			c[i*N+j]=0;
		}
	}
	hipMemcpy(dev_a,a,N*N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,N*N*sizeof(int),hipMemcpyHostToDevice);
	// cada bloque en dimensión x y y tendrá un tamaño de T Threads
	while((float)N/(float)div>32)
	{
		div++;
	}
	float f_N=(float)N,f_div=(float)div;
	T=(int)ceil(f_N/f_div);
	dim3 ThreadsBloque(T,T);
	// El grid tendrá B números de bloques en x y y
	dim3 Bloques(div, div);
	printf("Se va a realizar la suma con %d bloques y %d hilos\n",div,T);
	hipEventRecord(start,0);
	while(ind<iteraciones)
	{
		sumaMatrices<<<Bloques, ThreadsBloque>>>(dev_a,dev_b,dev_c,N);
		ind++;
	}
	
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	printf("El tiempo tomado para %d iteraciones fue de %3.3f ms\n",iteraciones,elapsedTime/10);
	hipMemcpy(c,dev_c,N*N*sizeof(int),hipMemcpyDeviceToHost);
	printf("Por ejemplo: \t%d\t+\t%d\t=%d\n",a[(int)N/2],b[(int)N/2],c[(int)N/2]);
	free(a);
	free(b);
	free(c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}