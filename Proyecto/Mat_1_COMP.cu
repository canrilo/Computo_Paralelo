
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define N 31
__global__ void matrixMultGPU(int *a, int *b, int *c){

	int k, sum = 0;	
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int fil = threadIdx.y + blockDim.y * blockIdx.y;
	__shared__ int s_a[N*N];
	__shared__ int s_b[N*N];
	if (col < N && fil < N) 
	{
		s_a[fil * N + col]=a[fil * N + col];
		s_b[fil * N + col]=b[fil * N + col];
		for (k = 0; k < N; k++) 
		{
			sum += s_a[fil * N + k] * s_b[k * N + col];
		}
		c[fil * N + col] = sum;
	}
}

int main (void){
	//Creación de variables del sistema
	int *a, *b, *c, *dev_a, *dev_b, *dev_c;
	int i,j;
	int T,div=1, iteraciones=100,ind=0;
	float elapsedTime;

	
	//Creación de variables de tiempo
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	printf("Creando espacio e inicializando matrices...\n");
	
	//Asignación e inicialización de memoria
	a=(int*)malloc(N*N*sizeof(int));
	b=(int*)malloc(N*N*sizeof(int));
	c=(int*)malloc(N*N*sizeof(int));
	
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
		{
			a[i*N+j]=i*j;
			b[i*N+j]=i*j;
		}
	}
	
	if(hipMalloc(&dev_a,N*N*sizeof(int))!=hipSuccess)
	{
		printf("########\nHubo un problema en la asignacion de memoria en la GPU\n########\n");
		exit(1);
	}
	if(hipMalloc(&dev_b,N*N*sizeof(int))!=hipSuccess)
	{
		printf("########\nHubo un problema en la asignacion de memoria en la GPU\n########\n");
		exit(1);
	}
	if(hipMalloc(&dev_c,N*N*sizeof(int))!=hipSuccess)
	{
		printf("########\nHubo un problema en la asignacion de memoria en la GPU\n########\n");
		exit(1);
	}
	
	printf("Asignacion de memoria correcta\n");
	//Copia de memoria a GPU
	if(hipMemcpy(dev_a,a,N*N*sizeof(int),hipMemcpyHostToDevice)!=hipSuccess)
	{
		printf("#########\nHubo un problema en la copia de memoria a la GPU\n#########\n");
		exit(1);
	}
	if(hipMemcpy(dev_b,b,N*N*sizeof(int),hipMemcpyHostToDevice)!=hipSuccess)
	{
		printf("#########\nHubo un problema en la copia de memoria a la GPU\n#########\n");
		exit(1);
	}
	
	//Cálculo de bloques e hilos
	while((float)N/(float)div>32)
	{
		div++;
	}
	float f_N=(float)N,f_div=(float)div;
	T=(int)ceil(f_N/f_div);
	dim3 ThreadsBloque(T,T);
	dim3 Bloques(div, div);
	printf("Se va a realizar la suma con %d bloques y %d hilos\n",div,T);
	printf("Se va a realizar %d iteraciones de matrices %dx%d\n",iteraciones,N,N);
	
	//Ejecución de kernel
	hipEventRecord(start,0);
	while(ind<iteraciones)
	{
		matrixMultGPU<<<Bloques, ThreadsBloque>>>(dev_a,dev_b,dev_c);
		ind++;
	}
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	
	printf("El tiempo tomado para %d iteraciones fue de %3.5f ms\n",iteraciones,elapsedTime);
	hipMemcpy(c,dev_c,N*N*sizeof(int),hipMemcpyDeviceToHost);
	printf("Por ejemplo %d deberia ser 0\n",c[3*N]);
	printf("Por ejemplo %d deberia ser 0\n",c[(int)N/2]);
	printf("Por ejemplo %d deberia ser %d\n",c[N+1],(int)((2*pow(N-1,3)+3*pow(N-1,2)+N-1)/6));
	/*
	for(i=0;i<N;i++)
	{	
		printf("\n");
		for(j=0;j<N;j++)
		{
			printf("\t%d",a[i*N+j]);
		}
		//printf("\t");
		for(j=0;j<N;j++)
		{
			printf("\t%d",b[i*N+j]);
		}
		//printf("\t");
		for(j=0;j<N;j++)
		{
			printf("\t%d",c[i*N+j]);
		}
	}
	*/
	
	free(a);
	free(b);
	free(c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	
	
	return 0;
}