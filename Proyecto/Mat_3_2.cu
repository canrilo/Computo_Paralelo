
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void matrixMultGPU(int *A,int *B,int *C, int N,int mod){

	int k, sum=0;
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int fil = threadIdx.y + blockDim.y * blockIdx.y;
	if (col < N && fil < N)
	{
		for (k = 0; k < N; k++) 
		{
			sum += A[fil * N + k] * B[k * N + col];
		}
		if(mod==0)
		{
			C[fil * N + col] = sum;	
		} else
		{
			C[fil * N + col] += sum;	
		}
		__syncthreads();
		
	}
}

int main (void){
	//Creación de variables del sistema
	int *a, *b, *c, N,NN;
	int *a_ul,*a_ur,*a_ll,*a_lr,*b_ul,*b_ur,*b_ll,*b_lr,*c_ul,*c_ur,*c_ll,*c_lr;
	int *DB,*DA,*DC1,*DC2;
	int i,j;
	int T,div=1, iteraciones=10,ind=0;
	float elapsedTime;
	printf("Ingrese el tamano deseado para las matrices:\n");
	scanf("%d",&NN);
	if(NN%2!=0 || NN<2)
	{
		printf("El tamaño debe ser mayor a dos y par\n");
		exit(1);
	}
	N=(int)NN/2;
	//Creación de variables de tiempo
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	printf("Creando espacio e inicializando matrices...\n");
	
	//Asignación e inicialización de memoria
	a=(int*)malloc(NN*NN*sizeof(int));
	b=(int*)malloc(NN*NN*sizeof(int));
	c=(int*)malloc(NN*NN*sizeof(int));
	
	a_ll=(int*)malloc(N*N*sizeof(int));
	a_lr=(int*)malloc(N*N*sizeof(int));
	a_ul=(int*)malloc(N*N*sizeof(int));
	a_ur=(int*)malloc(N*N*sizeof(int));
	
	b_ll=(int*)malloc(N*N*sizeof(int));
	b_lr=(int*)malloc(N*N*sizeof(int));
	b_ul=(int*)malloc(N*N*sizeof(int));
	b_ur=(int*)malloc(N*N*sizeof(int));
	
	c_ll=(int*)malloc(N*N*sizeof(int));
	c_lr=(int*)malloc(N*N*sizeof(int));
	c_ul=(int*)malloc(N*N*sizeof(int));
	c_ur=(int*)malloc(N*N*sizeof(int));
	
	//Inicialización de Matrices
	for(i=0;i<NN;i++)
	{
		for(j=0;j<NN;j++)
		{
			a[i*NN+j]=i*j;
			b[i*NN+j]=i*j;
		}
	}
	
	//Creación de submatrices
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
		{
			a_ul[i*N+j]=a[i*NN+j];
			a_ur[i*N+j]=a[i*NN+j+N];
			a_ll[i*N+j]=a[(i+N)*NN+j];
			a_lr[i*N+j]=a[(i+N)*NN+j+N];
			
			b_ul[i*N+j]=b[i*NN+j];
			b_ur[i*N+j]=b[i*NN+j+N];
			b_ll[i*N+j]=b[(i+N)*NN+j];
			b_lr[i*N+j]=b[(i+N)*NN+j+N];
		}
	}
	
	
	if(hipMalloc(&DA,N*N*sizeof(int))!=hipSuccess)
	{
		printf("########\nHubo un problema en la asignacion de memoria en la GPU\n########\n");
		exit(1);
	}
	if(hipMalloc(&DB,N*N*sizeof(int))!=hipSuccess)
	{
		printf("########\nHubo un problema en la asignacion de memoria en la GPU\n########\n");
		exit(1);
	}
	if(hipMalloc(&DC1,N*N*sizeof(int))!=hipSuccess)
	{
		printf("########\nHubo un problema en la asignacion de memoria en la GPU\n########\n");
		exit(1);
	}
	if(hipMalloc(&DC2,N*N*sizeof(int))!=hipSuccess)
	{
		printf("########\nHubo un problema en la asignacion de memoria en la GPU\n########\n");
		exit(1);
	}
	printf("Asignacion de memoria correcta\n");
	
	
	//Cálculo de bloques e hilos
	while((float)N/(float)div>32)
	{
		div++;
	}
	float f_N=(float)N,f_div=(float)div;
	T=(int)ceil(f_N/f_div);
	dim3 ThreadsBloque(T,T);
	dim3 Bloques(div, div);
	printf("Se va a realizar la suma con %d bloques y %d hilos\n",div,T);
	printf("Se va a realizar %d iteraciones de matrices %dx%d\n",iteraciones,NN,NN);
	
	//Ejecución de kernel
	hipEventRecord(start,0);
	while(ind<iteraciones)
	{	
		if(hipMemcpy(DA,a_ul,N*N*sizeof(int),hipMemcpyHostToDevice)!=hipSuccess)
		{
			printf("#########\nHubo un problema en la copia de memoria a la GPU\n#########\n");
			exit(1);
		}
		if(hipMemcpy(DB,b_ul,N*N*sizeof(int),hipMemcpyHostToDevice)!=hipSuccess)
		{
			printf("#########\nHubo un problema en la copia de memoria a la GPU\n#########\n");
			exit(1);
		}
		matrixMultGPU<<<Bloques, ThreadsBloque>>>(DA,DB,DC1,N,0);
		
		if(hipMemcpy(DB,b_ur,N*N*sizeof(int),hipMemcpyHostToDevice)!=hipSuccess)
		{
			printf("#########\nHubo un problema en la copia de memoria a la GPU\n#########\n");
			exit(1);
		}
		matrixMultGPU<<<Bloques, ThreadsBloque>>>(DA,DB,DC2,N,0);
		if(hipMemcpy(DA,a_ur,N*N*sizeof(int),hipMemcpyHostToDevice)!=hipSuccess)
		{
			printf("#########\nHubo un problema en la copia de memoria a la GPU\n#########\n");
			exit(1);
		}
		if(hipMemcpy(DB,b_ll,N*N*sizeof(int),hipMemcpyHostToDevice)!=hipSuccess)
		{
			printf("#########\nHubo un problema en la copia de memoria a la GPU\n#########\n");
			exit(1);
		}
		matrixMultGPU<<<Bloques, ThreadsBloque>>>(DA,DB,DC1,N,1);
		if(hipMemcpy(DB,b_lr,N*N*sizeof(int),hipMemcpyHostToDevice)!=hipSuccess)
		{
			printf("#########\nHubo un problema en la copia de memoria a la GPU\n#########\n");
			exit(1);
		}
		matrixMultGPU<<<Bloques, ThreadsBloque>>>(DA,DB,DC2,N,1);
		hipMemcpy(c_ul,DC1,N*N*sizeof(int),hipMemcpyDeviceToHost);
		hipMemcpy(c_ur,DC1,N*N*sizeof(int),hipMemcpyDeviceToHost);
		if(hipMemcpy(DA,a_ll,N*N*sizeof(int),hipMemcpyHostToDevice)!=hipSuccess)
		{
			printf("#########\nHubo un problema en la copia de memoria a la GPU\n#########\n");
			exit(1);
		}
		if(hipMemcpy(DB,b_ul,N*N*sizeof(int),hipMemcpyHostToDevice)!=hipSuccess)
		{
			printf("#########\nHubo un problema en la copia de memoria a la GPU\n#########\n");
			exit(1);
		}
		matrixMultGPU<<<Bloques, ThreadsBloque>>>(DA,DB,DC1,N,0);
		if(hipMemcpy(DB,b_ur,N*N*sizeof(int),hipMemcpyHostToDevice)!=hipSuccess)
		{
			printf("#########\nHubo un problema en la copia de memoria a la GPU\n#########\n");
			exit(1);
		}
		matrixMultGPU<<<Bloques, ThreadsBloque>>>(DA,DB,DC2,N,0);
		if(hipMemcpy(DA,a_lr,N*N*sizeof(int),hipMemcpyHostToDevice)!=hipSuccess)
		{
			printf("#########\nHubo un problema en la copia de memoria a la GPU\n#########\n");
			exit(1);
		}
		if(hipMemcpy(DB,b_ll,N*N*sizeof(int),hipMemcpyHostToDevice)!=hipSuccess)
		{
			printf("#########\nHubo un problema en la copia de memoria a la GPU\n#########\n");
			exit(1);
		}
		matrixMultGPU<<<Bloques, ThreadsBloque>>>(DA,DB,DC1,N,1);
		if(hipMemcpy(DB,b_lr,N*N*sizeof(int),hipMemcpyHostToDevice)!=hipSuccess)
		{
			printf("#########\nHubo un problema en la copia de memoria a la GPU\n#########\n");
			exit(1);
		}
		matrixMultGPU<<<Bloques, ThreadsBloque>>>(DA,DB,DC2,N,1);
		hipMemcpy(c_ll,DC1,N*N*sizeof(int),hipMemcpyDeviceToHost);
		hipMemcpy(c_lr,DC2,N*N*sizeof(int),hipMemcpyDeviceToHost);		
		
		ind++;
	}
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	printf("El tiempo tomado para %d iteraciones fue de %3.5f ms\n",iteraciones,elapsedTime);
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
		{
			c[i*NN+j]=c_ul[i*N+j];
			c[i*NN+j+N]=c_ur[i*N+j];
			c[(i+N)*NN+j]=c_ll[i*N+j];
			c[(i+N)*NN+j+N]=c_lr[i*N+j];
		}
	}
	
	printf("Por ejemplo %d deberia ser 0\n",c[3*NN]);
	printf("Por ejemplo %d deberia ser 0\n",c[(int)NN/2]);
	printf("Por ejemplo %d deberia ser %d\n",c[NN+1],(int)((2*pow(NN-1,3)+3*pow(NN-1,2)+NN-1)/6));
	/*
	for(i=0;i<NN;i++)
	{	
		printf("\n");
		for(j=0;j<NN;j++)
		{
			printf("\t%d",a[i*NN+j]);
		}
		//printf("\t");
		for(j=0;j<NN;j++)
		{
			printf("\t%d",b[i*NN+j]);
		}
		//printf("\t");
		for(j=0;j<NN;j++)
		{
			printf("\t%d",c[i*NN+j]);
		}
	}
	*/
	
	free(a);
	free(a_ll);
	free(a_lr);
	free(a_ul);
	free(a_ur);
	free(b_ur);
	free(b_ll);
	free(b_lr);
	free(b_ul);
	free(c_ll);
	free(c_lr);
	free(c_ul);
	free(c_ur);
	free(b);
	free(c);
	hipFree(DA);
	hipFree(DB);
	hipFree(DC1);
	hipFree(DC2);
		
	return 0;
}