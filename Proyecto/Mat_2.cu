
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void matrixMultGPU(int *a, int *b, int *c,int N)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	float sum=0;
	for (int k = 0; k < N; k++) 
	{
		float Ael= a[ty * N + k];
		float Bel= b[k * N + tx];
		sum += Ael*Bel;
	}
	c[ty * N + tx] = sum;
}


int main (void){
	//Creación de variables del sistema
	int *a, *b, *c, *dev_a, *dev_b, *dev_c, N;
	int i,j;
	int T,div=1, iteraciones=100,ind=0;
	float elapsedTime;
	printf("Ingrese el tamano deseado para las matrices:\n");
	scanf("%d",&N);
	
	//Creación de variables de tiempo
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	printf("Creando espacio e inicializando matrices...\n");
	
	//Asignación e inicialización de memoria
	a=(int*)malloc(N*N*sizeof(int));
	b=(int*)malloc(N*N*sizeof(int));
	c=(int*)malloc(N*N*sizeof(int));
	
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
		{
			a[i*N+j]=i*j;
			b[i*N+j]=i*j;
			c[i*N+j]=0;
		}
	}
	
	if(hipMalloc(&dev_a,N*N*sizeof(int))!=hipSuccess)
	{
		printf("########\nHubo un problema en la asignacion de memoria en la GPU\n########\n");
		exit(1);
	}
	if(hipMalloc(&dev_b,N*N*sizeof(int))!=hipSuccess)
	{
		printf("########\nHubo un problema en la asignacion de memoria en la GPU\n########\n");
		exit(1);
	}
	if(hipMalloc(&dev_c,N*N*sizeof(int))!=hipSuccess)
	{
		printf("########\nHubo un problema en la asignacion de memoria en la GPU\n########\n");
		exit(1);
	}
	
	printf("Asignacion de memoria correcta\n");
	//Copia de memoria a GPU
	if(hipMemcpy(dev_a,a,N*N*sizeof(int),hipMemcpyHostToDevice)!=hipSuccess)
	{
		printf("#########\nHubo un problema en la copia de memoria a la GPU\n#########\n");
		exit(1);
	}
	if(hipMemcpy(dev_b,b,N*N*sizeof(int),hipMemcpyHostToDevice)!=hipSuccess)
	{
		printf("#########\nHubo un problema en la copia de memoria a la GPU\n#########\n");
		exit(1);
	}
	
	//Cálculo de bloques e hilos
	while((float)N/(float)div>32)
	{
		div++;
	}
	float f_N=(float)N,f_div=(float)div;
	T=(int)ceil(f_N/f_div);
	//T=N;
	dim3 ThreadsBloque(T,T);
	//div=1;
	dim3 Bloques(div, div);
	printf("Se va a realizar la suma con %d bloques y %d hilos\n",div,T);
	printf("Se va a realizar %d iteraciones de matrices %dx%d\n",iteraciones,N,N);
	
	//Ejecución de kernel
	hipEventRecord(start,0);
	while(ind<iteraciones)
	{
		matrixMultGPU<<<Bloques, ThreadsBloque>>>(dev_a,dev_b,dev_c,N);
		ind++;
	}
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	
	printf("El tiempo tomado para %d iteraciones fue de %3.5f ms\n",iteraciones,elapsedTime);
	hipMemcpy(c,dev_c,N*N*sizeof(int),hipMemcpyDeviceToHost);
	printf("Por ejemplo %d deberia ser 0\n",c[3*N]);
	printf("Por ejemplo %d deberia ser 0\n",c[(int)N/2]);
	printf("Por ejemplo %d deberia ser %d\n",c[N+1],(int)((2*pow(N-1,3)+3*pow(N-1,2)+N-1)/6));
	
	/*for(i=0;i<N;i++)
	{	
		printf("\n");
		for(j=0;j<N;j++)
		{
			printf("\t%d",a[i*N+j]);
		}
		//printf("\t");
		for(j=0;j<N;j++)
		{
			printf("\t%d",b[i*N+j]);
		}
		//printf("\t");
		for(j=0;j<N;j++)
		{
			printf("\t%d",c[i*N+j]);
		}
	}
	*/	
	
	free(a);
	free(b);
	free(c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	
	
	return 0;
}