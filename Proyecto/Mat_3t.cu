
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void matrixMultGPU(int *a_ll,int *a_lr,int *a_ul, int *a_ur,int *b_ll,int *b_lr,int *b_ul, int *b_ur, int *c_ll,int *c_lr,int *c_ul, int *c_ur, int *t_ll,int *t_lr,int *t_ul, int *t_ur,int N){

	int k, sum_cur = 0,sum_cul = 0,sum_cll = 0,sum_clr = 0,sum_tur = 0,sum_tul = 0,sum_tll = 0,sum_tlr = 0;
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int fil = threadIdx.y + blockDim.y * blockIdx.y;
	if (col < N && fil < N) 
	{
		for (k = 0; k < N; k++) 
		{
			sum_cul += a_ul[fil * N + k] * b_ul[k * N + col];
			sum_cur += a_ul[fil * N + k] * b_ur[k * N + col];
			sum_cll += a_ll[fil * N + k] * b_ul[k * N + col];
			sum_clr += a_ll[fil * N + k] * b_ur[k * N + col];
			
			sum_tul += a_ur[fil * N + k] * b_ll[k * N + col];
			sum_tur += a_ur[fil * N + k] * b_lr[k * N + col];
			sum_tll += a_lr[fil * N + k] * b_ll[k * N + col];
			sum_tlr += a_lr[fil * N + k] * b_lr[k * N + col];
		}
		c_ul[fil * N + col] = sum_cul;
		c_ur[fil * N + col] = sum_cur;
		c_ll[fil * N + col] = sum_cll;
		c_lr[fil * N + col] = sum_clr;
		
		t_ul[fil * N + col] = sum_tul;
		t_ll[fil * N + col] = sum_tll;
		t_lr[fil * N + col] = sum_tlr;
		t_ur[fil * N + col] = sum_tur;
		__syncthreads();
		
		c_ul[fil * N + col]+=t_ul[fil * N + col];
		c_ll[fil * N + col]+=t_ll[fil * N + col];
		c_lr[fil * N + col]+=t_lr[fil * N + col];
		c_ur[fil * N + col]+=t_ur[fil * N + col];
	}
}

int main (void){
	//Creación de variables del sistema
	int *a, *b, *c, N,NN;
	int *a_ul,*a_ur,*a_ll,*a_lr,*b_ul,*b_ur,*b_ll,*b_lr,*c_ul,*c_ur,*c_ll,*c_lr;
	int *da_ul,*da_ur,*da_ll,*da_lr,*db_ul,*db_ur,*db_ll,*db_lr,*dc_ul,*dc_ur,*dc_ll,*dc_lr,*dt_ul,*dt_ur,*dt_ll,*dt_lr;
	int i,j;
	int T,div=1, iteraciones=10,ind=0;
	float elapsedTime;
	printf("Ingrese el tamano deseado para las matrices:\n");
	scanf("%d",&NN);
	if(NN%2!=0 || NN<2)
	{
		printf("El tamaño debe ser mayor a dos y par\n");
		exit(1);
	}
	N=(int)NN/2;
	//Creación de variables de tiempo
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	printf("Creando espacio e inicializando matrices...\n");
	
	//Asignación e inicialización de memoria
	a=(int*)malloc(NN*NN*sizeof(int));
	b=(int*)malloc(NN*NN*sizeof(int));
	c=(int*)malloc(NN*NN*sizeof(int));
	
	a_ll=(int*)malloc(N*N*sizeof(int));
	a_lr=(int*)malloc(N*N*sizeof(int));
	a_ul=(int*)malloc(N*N*sizeof(int));
	a_ur=(int*)malloc(N*N*sizeof(int));
	
	b_ll=(int*)malloc(N*N*sizeof(int));
	b_lr=(int*)malloc(N*N*sizeof(int));
	b_ul=(int*)malloc(N*N*sizeof(int));
	b_ur=(int*)malloc(N*N*sizeof(int));
	
	c_ll=(int*)malloc(N*N*sizeof(int));
	c_lr=(int*)malloc(N*N*sizeof(int));
	c_ul=(int*)malloc(N*N*sizeof(int));
	c_ur=(int*)malloc(N*N*sizeof(int));
	
	//Inicialización de Matrices
	for(i=0;i<NN;i++)
	{
		for(j=0;j<NN;j++)
		{
			a[i*NN+j]=i*j;
			b[i*NN+j]=i*j;
		}
	}
	
	//Creación de submatrices
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
		{
			a_ul[i*N+j]=a[i*NN+j];
			a_ur[i*N+j]=a[i*NN+j+N];
			a_ll[i*N+j]=a[(i+N)*NN+j];
			a_lr[i*N+j]=a[(i+N)*NN+j+N];
			
			b_ul[i*N+j]=b[i*NN+j];
			b_ur[i*N+j]=b[i*NN+j+N];
			b_ll[i*N+j]=b[(i+N)*NN+j];
			b_lr[i*N+j]=b[(i+N)*NN+j+N];
		}
	}
	
	{
	if(hipMalloc(&da_ll,N*N*sizeof(int))!=hipSuccess)
	{
		printf("########\nHubo un problema en la asignacion de memoria en la GPU\n########\n");
		exit(1);
	}
	if(hipMalloc(&da_ul,N*N*sizeof(int))!=hipSuccess)
	{
		printf("########\nHubo un problema en la asignacion de memoria en la GPU\n########\n");
		exit(1);
	}
	if(hipMalloc(&da_ur,N*N*sizeof(int))!=hipSuccess)
	{
		printf("########\nHubo un problema en la asignacion de memoria en la GPU\n########\n");
		exit(1);
	}
	if(hipMalloc(&da_lr,N*N*sizeof(int))!=hipSuccess)
	{
		printf("########\nHubo un problema en la asignacion de memoria en la GPU\n########\n");
		exit(1);
	}
	if(hipMalloc(&db_ll,N*N*sizeof(int))!=hipSuccess)
	{
		printf("########\nHubo un problema en la asignacion de memoria en la GPU\n########\n");
		exit(1);
	}
	if(hipMalloc(&db_lr,N*N*sizeof(int))!=hipSuccess)
	{
		printf("########\nHubo un problema en la asignacion de memoria en la GPU\n########\n");
		exit(1);
	}
	if(hipMalloc(&db_ul,N*N*sizeof(int))!=hipSuccess)
	{
		printf("########\nHubo un problema en la asignacion de memoria en la GPU\n########\n");
		exit(1);
	}
	if(hipMalloc(&db_ur,N*N*sizeof(int))!=hipSuccess)
	{
		printf("########\nHubo un problema en la asignacion de memoria en la GPU\n########\n");
		exit(1);
	}
	if(hipMalloc(&dc_ur,N*N*sizeof(int))!=hipSuccess)
	{
		printf("########\nHubo un problema en la asignacion de memoria en la GPU\n########\n");
		exit(1);
	}
	if(hipMalloc(&dc_ul,N*N*sizeof(int))!=hipSuccess)
	{
		printf("########\nHubo un problema en la asignacion de memoria en la GPU\n########\n");
		exit(1);
	}
	if(hipMalloc(&dc_ll,N*N*sizeof(int))!=hipSuccess)
	{
		printf("########\nHubo un problema en la asignacion de memoria en la GPU\n########\n");
		exit(1);
	}
	if(hipMalloc(&dc_lr,N*N*sizeof(int))!=hipSuccess)
	{
		printf("########\nHubo un problema en la asignacion de memoria en la GPU\n########\n");
		exit(1);
	}
	if(hipMalloc(&dt_ur,N*N*sizeof(int))!=hipSuccess)
	{
		printf("########\nHubo un problema en la asignacion de memoria en la GPU\n########\n");
		exit(1);
	}
	if(hipMalloc(&dt_ul,N*N*sizeof(int))!=hipSuccess)
	{
		printf("########\nHubo un problema en la asignacion de memoria en la GPU\n########\n");
		exit(1);
	}
	if(hipMalloc(&dt_ll,N*N*sizeof(int))!=hipSuccess)
	{
		printf("########\nHubo un problema en la asignacion de memoria en la GPU\n########\n");
		exit(1);
	}
	if(hipMalloc(&dt_lr,N*N*sizeof(int))!=hipSuccess)
	{
		printf("########\nHubo un problema en la asignacion de memoria en la GPU\n########\n");
		exit(1);
	}
	}
	printf("Asignacion de memoria correcta\n");
	//Cálculo de bloques e hilos
	while((float)N/(float)div>32)
	{
		div++;
	}
	float f_N=(float)N,f_div=(float)div;
	T=(int)ceil(f_N/f_div);
	dim3 ThreadsBloque(T,T);
	dim3 Bloques(div, div);
	printf("Se va a realizar la suma con %d bloques y %d hilos\n",div,T);
	printf("Se va a realizar %d iteraciones de matrices %dx%d\n",iteraciones,NN,NN);
	
	//Ejecución de kernel
	hipEventRecord(start,0);
	{
	//Copia de memoria a GPU
	if(hipMemcpy(da_ll,a_ll,N*N*sizeof(int),hipMemcpyHostToDevice)!=hipSuccess)
	{
		printf("#########\nHubo un problema en la copia de memoria a la GPU\n#########\n");
		exit(1);
	}
	if(hipMemcpy(da_lr,a_lr,N*N*sizeof(int),hipMemcpyHostToDevice)!=hipSuccess)
	{
		printf("#########\nHubo un problema en la copia de memoria a la GPU\n#########\n");
		exit(1);
	}
	if(hipMemcpy(da_ul,a_ul,N*N*sizeof(int),hipMemcpyHostToDevice)!=hipSuccess)
	{
		printf("#########\nHubo un problema en la copia de memoria a la GPU\n#########\n");
		exit(1);
	}
	if(hipMemcpy(da_ur,a_ur,N*N*sizeof(int),hipMemcpyHostToDevice)!=hipSuccess)
	{
		printf("#########\nHubo un problema en la copia de memoria a la GPU\n#########\n");
		exit(1);
	}
	if(hipMemcpy(db_ll,b_ll,N*N*sizeof(int),hipMemcpyHostToDevice)!=hipSuccess)
	{
		printf("#########\nHubo un problema en la copia de memoria a la GPU\n#########\n");
		exit(1);
	}
	if(hipMemcpy(db_lr,b_lr,N*N*sizeof(int),hipMemcpyHostToDevice)!=hipSuccess)
	{
		printf("#########\nHubo un problema en la copia de memoria a la GPU\n#########\n");
		exit(1);
	}
	if(hipMemcpy(db_ul,b_ul,N*N*sizeof(int),hipMemcpyHostToDevice)!=hipSuccess)
	{
		printf("#########\nHubo un problema en la copia de memoria a la GPU\n#########\n");
		exit(1);
	}
	if(hipMemcpy(db_ur,b_ur,N*N*sizeof(int),hipMemcpyHostToDevice)!=hipSuccess)
	{
		printf("#########\nHubo un problema en la copia de memoria a la GPU\n#########\n");
		exit(1);
	}
	}
	
	
	while(ind<iteraciones)
	{	
		matrixMultGPU<<<Bloques, ThreadsBloque>>>(da_ll,da_lr,da_ul,da_ur,db_ll,db_lr,db_ul,db_ur,dc_ll,dc_lr,dc_ul,dc_ur,dt_ll,dt_lr,dt_ul,dt_ur,N);
		ind++;
	}
	
	
	
	hipMemcpy(c_ll,dc_ll,N*N*sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(c_lr,dc_lr,N*N*sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(c_ur,dc_ur,N*N*sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(c_ul,dc_ul,N*N*sizeof(int),hipMemcpyDeviceToHost);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	printf("El tiempo tomado para %d iteraciones fue de %3.5f ms\n",iteraciones,elapsedTime);
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
		{
			c[i*NN+j]=c_ul[i*N+j];
			c[i*NN+j+N]=c_ur[i*N+j];
			c[(i+N)*NN+j]=c_ll[i*N+j];
			c[(i+N)*NN+j+N]=c_lr[i*N+j];
		}
	}
	
	printf("Por ejemplo %d deberia ser 0\n",c[3*NN]);
	printf("Por ejemplo %d deberia ser 0\n",c[(int)NN/2]);
	printf("Por ejemplo %d deberia ser %d\n",c[NN+1],(int)((2*pow(NN-1,3)+3*pow(NN-1,2)+NN-1)/6));
	/*
	for(i=0;i<NN;i++)
	{	
		printf("\n");
		for(j=0;j<NN;j++)
		{
			printf("\t%d",a[i*NN+j]);
		}
		//printf("\t");
		for(j=0;j<NN;j++)
		{
			printf("\t%d",b[i*NN+j]);
		}
		//printf("\t");
		for(j=0;j<NN;j++)
		{
			printf("\t%d",c[i*NN+j]);
		}
	}
	*/
	
	free(a);
	free(a_ll);
	free(a_lr);
	free(a_ul);
	free(a_ur);
	free(b_ur);
	free(b_ll);
	free(b_lr);
	free(b_ul);
	free(c_ll);
	free(c_lr);
	free(c_ul);
	free(c_ur);
	free(b);
	free(c);
	hipFree(da_ll);
	hipFree(da_lr);
	hipFree(da_ul);
	hipFree(da_ur);
	hipFree(db_ll);
	hipFree(db_lr);
	hipFree(db_ul);
	hipFree(db_ur);
	hipFree(dc_ll);
	hipFree(dc_lr);
	hipFree(dc_ul);
	hipFree(dc_ur);
	hipFree(dt_ll);
	hipFree(dt_lr);
	hipFree(dt_ul);
	hipFree(dt_ur);
		
	return 0;
}